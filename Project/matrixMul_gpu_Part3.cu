#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>

__global__
void GPUmatmul(double *x, double *y, double *ans, int N) 
{
  //calculates the unique thread ID in the block
	int t= (blockDim.x*blockDim.y)*threadIdx.z+(threadIdx.y*blockDim.x)+(threadIdx.x);
	//calculates the unique block ID in the grid
	int b= (gridDim.x*gridDim.y)*blockIdx.z+(blockIdx.y*gridDim.x)+(blockIdx.x);
	//block size 
	int T= blockDim.x*blockDim.y*blockDim.z;
	//grid size
	int B= gridDim.x*gridDim.y*gridDim.z;
	 
    for (int i=b;i<N;i+=B)
    {
		for(int j=t;j<N;j+=T)
		{
			for(int k=0;k<N;k++)
			{
				ans[i*N+j]+=(x[i*N+k]*y[k*N+j]);
			}
		}
	}
}

// ---------------------------------------------------------------------- check
bool check(int N, double *ans)
{
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      if(ans[i*N+j] != 20.0) return false;
    }
  }
  return true;
}

int main(int argc, char const *argv[])
{
    int N= 1<<9;
    int iter = 3;
    clock_t t;

    // matrices
    double *x, *y, *ans;
	
    //allocate memory -- accessible from both GPU and CPU
    hipMallocManaged((void **) &x, N*N*sizeof(double));
    hipMallocManaged((void **) &y, N*N*sizeof(double));
    hipMallocManaged((void **) &ans, N*N*sizeof(double));

    // random initialize matrix A
    for (int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            x[i*N+j] = 5;
            y[i*N+j] = (i==j?1:0);
            ans[i*N+j] = (double)0.000000000000;
        }
    }
    
    // Run the kernel
    double avg = 0;
    std::cout<<"Starting GPU computation"<<std::endl;
    for(int i = 0; i <= iter; i++) {
        t = clock();
        GPUmatmul<<<dim3(16,4,4), dim3(8,8,8)>>>(x, y, ans, N);
        t = clock() - t;
        if(i) avg += t;
    }

    avg /= iter;
    avg /= CLOCKS_PER_SEC;
    avg *= 1000;
    printf("It took %lf ms on avg.\n", avg);
    hipDeviceSynchronize();

    // validate results computed by GPU
    if(check(N,ans)) std::cout<<"RUN OK."<<std::endl;
        else std::cout<<"RUN NOT OK."<<std::endl;

    // free memory
    hipHostFree(x);
    hipHostFree(y);
    hipHostFree(ans);
    return 0;
}
